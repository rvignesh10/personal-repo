#include "hip/hip_runtime.h"
#include <stdio.h>

#include <ctime>
// ------------------------------------------------------------------------
// Return the current wall-clock time in seconds
// ------------------------------------------------------------------------
inline double getCPU(){
    return ( 1.0 * std::clock() )/CLOCKS_PER_SEC ;
}

#include "parseCommand.h"

__global__ void add( int *a_d, int *b_d, int *c_d, int *n_d, int *nt_d ){
  int idx= threadIdx.x + blockIdx.x*(*nt_d);
  if(idx < *n_d)
    c_d[idx] = a_d[idx] + b_d[idx];
}

int main( int argc, char *argv[] ) {

  int n=1000;

  std::string line;
  bool echo=true;
  for( int i=1; i<argc; i++ )
  {
    line=argv[i];
    if( parseCommand( line, "-n=", n, echo ) ){}
  }

  int *a_p = new int [n];
  int *b_p = new int [n];
  int *c_p = new int [n];

  double cpu0 = getCPU();
  for( int i=0; i<n; i++ ) {
    a_p[i] = -i; b_p[i] = i*i;
    c_p[i] = a_p[i] + b_p[i];
  }
  double cpuTime= getCPU() - cpu0;
  int *a_d;
  int *b_d;
  int *c_d;
  int *n_d;
  int *nt_d;
  hipMalloc((void **)&c_d, n*sizeof(int));
  hipMalloc((void **)&a_d, n*sizeof(int));
  hipMalloc((void **)&b_d, n*sizeof(int));
  hipMalloc((void **)&n_d, sizeof(int));
  hipMalloc((void **)&nt_d,sizeof(int));
  hipMemcpy(a_d, a_p, n*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_p, n*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(n_d, &n ,   sizeof(int), hipMemcpyHostToDevice);
  //hipMemcpy(nt_d,&Nt,   sizeof(int), hipMemcpyHostToDevice);
  int Nt;
  for( int k=0; k<=10; k++ ){
    Nt=(int)(pow(2, k));
    int Nb = ceil( (1.*n)/Nt );
    hipMemcpy(nt_d,&Nt,   sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    add<<<Nb, Nt>>>(a_d, b_d, c_d, n_d, nt_d);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float gpuTime;
    hipEventElapsedTime(&gpuTime, start, stop);

    hipMemcpy(c_p, c_d, n*sizeof(int), hipMemcpyDeviceToHost);

    double maxErr=0.0;
    for(int i=0; i<n; i++){
      double t = fabs(c_p[i]- a_p[i]-b_p[i]);
      maxErr = t>=maxErr? t:maxErr;
    }
    // printf("maxErr=%f\n",maxErr); 
   printf("%10d \t %10d \t %10d \t %10d \t %25.15e \t %25.15e \t %f\n", Nb, Nt, Nb*Nt, n, maxErr, gpuTime, cpuTime*1000/gpuTime);   
  }
  hipFree(a_d); hipFree(b_d);
  hipFree(c_d); hipFree(n_d); hipFree(nt_d);
  delete [] a_p;
  delete [] b_p;
  delete [] c_p;
  
  return 0;
}