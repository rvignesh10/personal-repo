#include "hip/hip_runtime.h"
/* multiply.cu */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

const double b0 = 1.0, b1 = 0.5, b2 = 0.25;
const double a0 = 1.0, a1 = 0.3;

#define UTRUE(x, t) ( b0 + (x) * (b1 + (x) * b2) ) * ( a0 + (t) * a1 )
#define UTRUEX(x, t) ( b1 + 2. * (x) * b2 ) * ( a0 + (t) * ( a1 ) )
#define UTRUET(x, t) (b0 + (x) * ( b1 + (x) * b2 )) * ( a1 )
#define UTRUEXX(x, t) ( 2.*b2 )*( a0 + (t)*( a1 ) )
#define FORCE(x, t) ( UTRUET(x, t) - kappa*UTRUEXX(x, t) )

__global__ void __setSpatialMeshNodes1D__(double *dev_x, double *dev_xa, double *dev_dx, int *dev_nd1L, int *dev_Nt)
{
    const int idx = threadIdx.x + blockIdx.x * (*dev_Nt);
    if( idx < *dev_nd1L )
        dev_x[idx] = *dev_xa + (idx)*(*dev_dx);
}

__global__ void __heat1dSetInitialCondition__ (double *dev_uc, double *dev_x, int *dev_Nt, int *dev_nd1L) 
{
    const int idx = threadIdx.x + blockIdx.x * (*dev_Nt);
    if( idx < *dev_nd1L )
        dev_uc[idx] = UTRUE( dev_x[idx], 0. );
}

__global__ void __heat1dForwardEulerUpdate__ (double *dev_uc, double *dev_un, double *dev_x, double *dev_rx, double *dev_t, int *dev_Nt, int *dev_n1aL, int *dev_n1bL, int *dev_numGhost)
{
    // const int idx = threadIdx.x + blockIdx.x * blockDim.x;
    const int idx = threadIdx.x + blockIdx.x * (*dev_Nt);
    // check if idx is within the internal range
    if( (idx>= *dev_n1aL + *dev_numGhost) || (idx <= *dev_n1bL + *dev_numGhost) )
        dev_un[idx] = (*dev_rx)*(dev_uc[idx+1] -2.*dev_uc[idx] + dev_uc[idx-1]) + FORCE( dev_x[idx], *dev_t ) ;
}

__global__ void __heat1dErrorCalc__(double *dev_err, double *dev_uc, double *dev_x, double *dev_t, int *dev_nd1L, int *dev_Nt)
{
    const int idx = threadIdx.x + blockIdx.x * (*dev_Nt);
    if(idx < *dev_nd1L)
        dev_err[idx] = dev_uc[idx] - UTRUE( dev_x[idx], *dev_t );
}

extern "C" void setSpatialMeshNodes1D( double *dev_x, double *dev_xa, double *dev_dx, int *dev_nd1L, int Nb, int Nt, int *dev_Nt )
{
    __setSpatialMeshNodes1D__<<<Nb, Nt>>>(dev_x, dev_xa, dev_dx, dev_nd1L, dev_Nt);
    safecall(hipDeviceSynchronize());
    safecall(hipGetLastError());
}

extern "C" void heat1dSetInitialCondition(double *dev_uc, double *dev_x, int Nb, int Nt, int *dev_Nt, int *dev_nd1L)
{   
    __heat1dSetInitialCondition__<<<Nb, Nt>>> (dev_uc, dev_x, dev_Nt, dev_nd1L);
    safecall(hipDeviceSynchronize());
    safecall(hipGetLastError());
}

extern "C" void heat1dForwardEulerUpdate(double *dev_uc, double *dev_un, double *dev_x, double *dev_rx, double *dev_t, 
                                         int Nb, int Nt, int *dev_Nt, int *dev_n1aL, int *dev_n1bL, int *dev_numGhost, float *gpuStepTime)
{   
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    __heat1dForwardEulerUpdate__ <<<Nb, Nt>>> ( dev_uc, dev_un, dev_x, dev_rx, dev_t, dev_Nt, dev_n1aL, dev_n1bL, dev_numGhost );
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(gpuStepTime, start, stop);

    safecall(hipDeviceSynchronize());
    safecall(hipGetLastError());
}

extern "C" void heat1dErrorCalc(double *dev_err, double *dev_uc, double *dev_x, double *dev_t, int Nb, int Nt, int *dev_Nt, int *dev_nd1L)
{
    __heat1dErrorCalc__<<<Nb, Nt>>>(dev_err, dev_uc, dev_x, dev_t, dev_nd1L, dev_Nt);
    safecall(hipDeviceSynchronize());
    safecall(hipGetLastError());
}

/* --------------------------------------------------------------------------------------------------- */
// wrapper functions for allocating and free-ing cuda memory 
/* --------------------------------------------------------------------------------------------------- */
extern "C" void AllocateCudaMemory( double *dev_var, int n_size ) { hipMalloc((void **)&dev_var, (n_size)*sizeof(double)); }
extern "C" void AllocateCudaMemory( int *dev_var, int n_size ) { hipMalloc((void **)&dev_var, (n_size)*sizeof(int)); }

extern "C" void FreeCudaMemory( double *dev_var ) { hipFree(dev_var); }
extern "C" void FreeCudaMemory( int *dev_var ) { hipFree(dev_var); }

/* --------------------------------------------------------------------------------------------------- */
// wrapper functions for copying memory
/* --------------------------------------------------------------------------------------------------- */
extern "C" void MemcpyHostToDev( double *dev_var, double *hst_var, int n_size ) { hipMemcpy( dev_var, hst_var, (n_size)*sizeof(double), hipMemcpyHostToDevice ); }
extern "C" void MemcpyHostToDev( int *dev_var, int *hst_var, int n_size ) { hipMemcpy( dev_var, hst_var, (n_size)*sizeof(int), hipMemcpyHostToDevice ); }

extern "C" void MemcpyDevToHost( double *hst_var, double *dev_var, int n_size ) { hipMemcpy( hst_var, dev_var, (n_size)*sizeof(double), hipMemcpyDeviceToHost ); }
extern "C" void MemcpyDevToHost( int *hst_var, int *dev_var, int n_size ) { hipMemcpy( hst_var, dev_var, (n_size)*sizeof(int), hipMemcpyDeviceToHost ); }
