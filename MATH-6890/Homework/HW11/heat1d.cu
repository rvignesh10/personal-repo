#include "hip/hip_runtime.h"
/*  ------------------------- Solving Heat Equation in 1D ------------------------- */

// ------------------------------------------------------------------------
// ways to execute this code 
// ./heat1d -Nx #discretizationSize -matlabFileName #fileName.m
// examples:
// ./heat1d -Nx 40 -matlabFileName heat1dBcDDNx20.m
// ------------------------------------------------------------------------

#include <stdio.h>
#include <iostream>
#include <math.h>
#include <float.h>
#include <assert.h>

// define a new type "Real" which is equivalent to double
typedef double Real;
#define MAX_NUM_THREADS 128

#include <string>
using std::string;
using std::max;

#include <ctime>


// some necessary constants
Real kappa  = 0.1;
const Real kx = 3.0;
const Real kxpi = kx * M_PI;
const Real kappaPiSq = kappa * kxpi * kxpi;

const Real b0 = 1.0, b1 = 0.5, b2 = 0.25;
const Real a0 = 1.0, a1 = 0.3;

// defining different type of solutions available in the code
#define TRIG_DD 1
#define TRIG_NN 2
#define POLY_DD 3
#define POLY_NN 4

#ifndef SOLUTION
    #define SOLUTION TRIG_DD
    // #define SOLUTION TRIG_NN
    // #define SOLUTION POLY_DD
    // #define SOLUTION POLY_NN
#endif

// assign boundary conditions for different problems
#if SOLUTION == TRIG_DD
    const char solutionName[] = "trueDD";

    #define UTRUE(x, t) sin( kxpi * (x) ) * exp( -kappaPiSq * (t) )
    #define UTRUEX(x, t) kxpi * cos( kxpi * (x) ) * exp( -kappaPiSq * (t) )
    #define UTRUET(x, t) -kappaPiSq * UTRUE(x, t)
    #define FORCE(x, t) (0.)

#elif SOLUTION == TRIG_NN
    const char solutionName[] = "trueNN";

    #define UTRUE(x, t) cos( kxpi * (x) ) * exp( -kappaPiSq * (t) )
    #define UTRUEX(x, t)  -kxpi * sin( kxpi * (x) ) * exp( -kappaPiSq * (t) )
    #define UTRUET(x, t) -kappaPiSq * UTRUE(x, t)
    #define FORCE(x, t) (0.)

#elif SOLUTION == POLY_DD || SOLUTION == POLY_NN
    #if SOLUTION == POLY_DD
        const char solutionName[] = "polyDD";
    #else
        const char solutionName[] = "polyNN";
    #endif

    #define UTRUE(x, t) ( b0 + (x) * (b1 + (x) * b2) ) * ( a0 + (t) * a1 )
    #define UTRUEX(x, t) ( b1 + 2. * (x) * b2 ) * ( a0 + (t) * ( a1 ) )
    #define UTRUET(x, t) (b0 + (x) * ( b1 + (x) * b2 )) * ( a1 )
    #define UTRUEXX(x, t) ( 2.*b2 )*( a0 + (t)*( a1 ) )
    #define FORCE(x, t) ( UTRUET(x, t) - kappa*UTRUEXX(x, t) )
#else
    const char solutionName[] = "wrongChoice";
#endif

/* --------------------------------------------------------------------- */
//                              cuda kernels
/* --------------------------------------------------------------------- */

__global__ void mesh1d(double *x_d, int *nd1_d, int *nd1a_d, int *nd1b_d, double *xa_d, double *dx_d, int *nt_d){
    const int idx = threadIdx.x + blockIdx.x * (*nt_d) + *nd1a_d ;
    #define x(i1)  x_d[i1 - *nd1a_d]
    if (idx >= *nd1a_d && idx <= *nd1b_d)
        x(idx) = *xa_d + idx*(*dx_d);
    #undef x
}

__global__ void setInitialCondition(double *u_d, int *nd1_d, int *nd1a_d, int *nd1b_d, double *x_d, double *t_d, int *nt_d){
    const int idx = threadIdx.x + blockIdx.x * (*nt_d) + *nd1a_d ;
    #define u(i1) u_d[i1 - *nd1a_d]
    #define x(i1) x_d[i1 - *nd1a_d]

    if(idx >= *nd1a_d && idx <= nd1b_d)
        u(idx) = UTRUE( x(idx), *t_d );

    #undef x
    #undef u
}

__global__ void heat1dForwardEulerTimeStep( double *uc_d, double *un_d, double *x_d,
                                            int *nd1_d, int *nd1a_d, int *nd1b_d, 
                                            double *rx_d, double *dt_d, double *t_d, int *nt_d ){
    const int idx = threadIdx.x + blockIdx.x * (*nt_d) + *nd1a_d ;

    #define uc(i1) uc_d[i1- *nd1a_d]
    #define un(i1) un_d[i1- *nd1a_d]
    #define x(i1)  x_d[i1 - *nd1a_d]

    if(idx > *nd1a_d && idx < *nd1b_d)
    {
        un(idx) = uc(idx) + (*rx_d) * ( uc(idx-1) + 2.*uc(idx) + uc(idx+1) ) + 
                  (*dt_d) * FORCE( x(idx), *t_d );
    }

    #undef uc
    #undef un
    #undef x
}


// ------------------------------------------------------------------------
// Return the current wall-clock time in seconds
// ------------------------------------------------------------------------
inline double getCPU(){
    return ( 1.0 * std::clock() )/CLOCKS_PER_SEC ;
}

#include "parseCommand.h"

// ------------------------------------------------------------------------
// Function to save a vector to a matlab file
// matlabFile  (input) : save vector to this file
// u_p         (input) : array of vector values
// name        (input) : name for array
// (nd1a:nd1b) (input) : array dimensions
// ------------------------------------------------------------------------
int writeMatlabVector(FILE *matlabFile, Real *u_p, const char *name, int nd1a, int nd1b){
    #define u(i) u_p[i-nd1a]

    const int numPerLine=8; // number of entries per line
    // Save the vector as:
    // name = [ num num num num num ...
    // num num num num num ];
    fprintf(matlabFile,"%s=[",name);
    for( int i=nd1a; i<=nd1b; i++ ) {
        fprintf(matlabFile,"%20.15e ",u(i));
        if( (i-nd1a) % numPerLine == numPerLine-1 )
            fprintf(matlabFile,"...\n"); // continuation line
    }
    fprintf(matlabFile,"];\n");

    return 0;
}


int main(int argc, char *argv[]){

    // parameters and names used if it is not provided 
    int Nx = 20;           // set spatial discretization size
    std::string matlabFileName = "heat1d_gpu.m";
    int debug= 0;
    int saveMatlab=0;
    Real tFinal = 0.2;
    Real cfl    = 0.9;

    std::string line;
    bool echo = true;
    for (int i=1; i<argc; i++)
    {
        line = argv[i];
        if( parseCommand(line, "-nx=", Nx, echo) ) {}
        else if( parseCommand(line, "-debug=", debug, echo) ) {}
        else if( parseCommand(line, "-tFinal=", tFinal, echo) ) {}
        else if( parseCommand(line, "-saveMatlab=", saveMatlab, echo) ) {}
        else if( parseCommand(line, "-matlabFileName=", matlabFileName, echo) ) {}
    }
    
    Real xa = 0.0, xb = 1.0 ;
    Real *xa_d;
    checkCudaErrors( hipMalloc((void **)&xa_d, sizeof(Real)) ); hipMemcpy(xa_d, &xa, sizeof(Real), hipMemcpyHostToDevice);

    // ============= Grid and indexing==============
    //            xa                             xb
    //         G---X---+---+---+---+-- ... ---+---X---G
    //             0   1   2                      Nx
    //             ja                             jb
    //        nd1a                                   nd1b
    // C index: 0 1 2 3 ...

    Real dx = (xb - xa)/Nx;
    Real *dx_d;
    checkCudaErrors( hipMalloc((void **)&dx_d, sizeof(Real)) ); hipMemcpy(dx_d, &dx, sizeof(Real), hipMemcpyHostToDevice);
    int numGhost = 1;
    int n1a       = 0;
    int n1b       = Nx;
    int nd1a     = n1a - numGhost;
    int nd1b     = n1b + numGhost;
    int nd1      = nd1b - nd1a + 1; // total number of grid points including ghost nodes

    int nt = MAX_NUM_THREADS;
    int nb = ceil( (1.*nd1)/nt );
    int *nt_d;
    checkCudaErrors( hipMalloc((void **)&nt_d, sizeof(int)) );  hipMemcpy(nt_d, &nt, sizeof(int), hipMemcpyHostToDevice);

    int *n1a_d, *n1b_d, *nd1a_d, *nd1b_d, *nd1_d;
    checkCudaErrors( hipMalloc((void **)&n1a_d, sizeof(int)) );  hipMemcpy(n1a_d, &n1a, sizeof(int), hipMemcpyHostToDevice);
    checkCudaErrors( hipMalloc((void **)&n1b_d, sizeof(int)) );  hipMemcpy(n1b_d, &n1b, sizeof(int), hipMemcpyHostToDevice);
    checkCudaErrors( hipMalloc((void **)&nd1a_d, sizeof(int)) ); hipMemcpy(nd1a_d,&nd1a,sizeof(int), hipMemcpyHostToDevice);
    checkCudaErrors( hipMalloc((void **)&nd1b_d, sizeof(int)) ); hipMemcpy(nd1b_d,&nd1b,sizeof(int), hipMemcpyHostToDevice);
    checkCudaErrors( hipMalloc((void **)&nd1_d, sizeof(int)) );  hipMemcpy(nd1_d, &nd1, sizeof(int), hipMemcpyHostToDevice);
    // creating a 1D array of grid points
    Real *x_p = new Real [nd1];
    # define x(i) x_p[i-nd1a] 
    Real *x_d;
    cudaCheckErrors( hipMalloc((void **)&x_d, nd1*sizeof(Real)) ); 
    mesh1d<<<nb, nt>>>(x_d, nd1_d, nd1a_d, nd1b_d, xa_d, dx_d, nt_d);
    hipMemcpy(x_p, x_d, nd1*sizeof(Real), hipMemcpyDeviceToHost);

    const int numSides = 2;
    const int dirichlet = 1, neumann = 2;
    const int numberOfDimensions = 1;
    // initialize boundary conditions as a 2D matrix with rows for left/right side and columns for dimensions
    int *boundaryCondition_p = new int [numSides * numberOfDimensions];
    #define boundaryCondition(side, axis) boundaryCondition_p[(side) + numSides*(axis)]

    // assign boundary conditions for different problems
    #if SOLUTION == TRIG_DD
        // true solution for dirichlet BC's
        boundaryCondition(0, 0) = dirichlet;
        boundaryCondition(1, 0) = dirichlet;
    
    #elif SOLUTION == TRIG_NN
        // true solution for neumann BC's
        boundaryCondition(0, 0) = neumann;
        boundaryCondition(1, 0) = neumann;

    #elif SOLUTION == POLY_DD || SOLUTION == POLY_NN
        #if SOLUTION == POLY_DD
            boundaryCondition(0, 0) = dirichlet;
            boundaryCondition(1, 0) = dirichlet;

        #else
            boundaryCondition(0, 0) = neumann;
            boundaryCondition(1, 0) = neumann;
        
        #endif
    #else
        std::cerr << "ERROR: unknown choice of solution/case to solve \n";
        abort();
    #endif

    Real *u_p[2]; // two arrays used for storing current and next time step solution vectors
    u_p[0] = new Real [nd1];
    u_p[1] = new Real [nd1];
    
    #define uc(i) u_p[curr][i-nd1a]
    #define un(i) u_p[next][i-nd1a]

    // initial condition set up
    Real t = 0.0;
    Real *t_d;
    checkCudaErrors( hipMalloc((void **)&t_d, sizeof(Real)) ); hipMemcpy(t_d, &t, sizeof(Real), hipMemcpyHostToDevice);
    int curr = 0;
    Real *u_d[2];
    checkCudaErrors( hipMalloc(void **)&u_d[0], nd1*sizeof(Real) );
    checkCudaErrors( hipMalloc(void **)&u_d[1], nd1*sizeof(Real) );
    Real *uc_h = &uc(nd1a);
    Real *un_h = nullptr;
    Real *uc_d = u_d+curr;
    Real *un_d = nullptr;
    setInitialCondition(uc_d, nd1_d, nd1a_d, nd1b_d, x_d, t_d, nt_d);
    hipMemcpy(uc_h, uc_d, nd1*sizeof(Real), hipMemcpyDeviceToHost);

    /* Time-step restrictions */
    const Real dx2     = dx * dx;
    Real dt            = cfl * 0.5 * dx2 / kappa;
    const int numSteps = ceil( tFinal/dt );
    dt                 = tFinal/numSteps;
    Real *dt_d;
    checkCudaErrors( hipMalloc((void **)&dt_d, sizeof(Real)) ); hipMemcpy(dt_d, &dt, sizeof(Real), hipMemcpyHostToDevice);
    const Real rx      = kappa * dt / dx2;
    Real *rx_d;
    checkCudaErrors( hipMalloc((void **)&rx_d, sizeof(Real)) ); hipMemcpy(rx_d, &rx, sizeof(Real), hipMemcpyHostToDevice);

    printf("------------------- Solve the heat equation in 1D solution=%s --------------------- \n",solutionName);
    printf("  numGhost=%d, n1a=%d, n1b=%d, nd1a=%d, nd1b=%d\n",numGhost,ja,jb,nd1a,nd1b);
    printf("  numSteps=%d, Nx=%d, kappa=%g, tFinal=%g, boundaryCondition(0,0)=%d, boundaryCondition(1,0)=%d\n",numSteps,Nx,kappa,tFinal,boundaryCondition(0,0),boundaryCondition(1,0));
    printf(" ----------------- Using Forward Euler Time Stepping -------------------------- \n");

    /* -------- TIME-STEPPING LOOP --------- */
    Real cpu0 = getCPU();
    for (int n=0; n<numSteps; n++) {
        const int curr = n % 2;
        const int next = (n + 1) % 2;
        t = n * dt; // current time
        hipMemcpy(t_d, &t, sizeof(Real), hipMemcpyHostToDevice);
        
        uc_h = &uc(nd1a);
        un_h = &un(nd1a);

        uc_d = u_d+curr;
        un_d = u_d+next;
        hipMemcpy(uc_d, uc_h, nd1*sizeof(Real), hipMemcpyHostToDevice);
        heat1dForwardEulerTimeStep<<<nb, nt>>>(uc_d, un_d, x_d, nd1_d, nd1a_d, nd1b_d, rx_d, dt_d, t_d, nt_d);
        hipMemcpy(un_h, un_d, nd1*sizeof(Real), hipMemcpyDeviceToHost);
        // set boundary conditions on host
        for (int side=0; side<=1; side++) {
            const int i  = side == 0 ? ja : jb;
            const int is = 1 - 2*side;
            if (boundaryCondition(side, 0) == dirichlet){
                un(i)    = UTRUE( x(i), t+dt );
                un(i-is) = 3.0 * un(i) - 3.0 * un(i+is) + un(i+2*is); // extrapolate ghost
            }
            else {
                // neumann
                un(i-is) = un(i+is) - 2.*is*dx*UTRUEX(x(i),t+dt);
            }
        }

    }
    Real cpuTimeStep = getCPU()-cpu0;

    /*  check errors */
    t += dt; // tFinal
    if ( fabs(t - tFinal) > 1e-3 * dt/tFinal ){
        printf("ERROR AFTER TIME STEPPING: t=%16.8e IS NOT EQUAL to tFinal=%16.8e\n", t, tFinal);
    }

    Real *error_p = new Real [nd1];
    #define error(i) error_p[i-nd1a]

    curr = numSteps % 2;
    Real maxErr = 0.0;
    for (int i=nd1a; i<=nd1b; i++) {
        error(i) = uc(i) - UTRUE( x(i), t );
        maxErr = max( maxErr, abs(error(i)) );
    }
    printf("numSteps=%4d, Nx=%3d, maxErr=%9.2e, cpu=%9.2e(s)\n",numSteps,Nx,maxErr,cpuTimeStep);

// --- Write a file for plotting in matlab ---
    FILE *matlabFile = fopen(matlabFileName.c_str(),"w");
    fprintf(matlabFile,"%% File written by heat1d.C\n");
    fprintf(matlabFile,"xa=%g; xb=%g; kappa=%g; t=%g; maxErr=%10.3e; cpuTimeStep=%10.3e;\n",xa,xb,kappa,tFinal,maxErr,cpuTimeStep);
    fprintf(matlabFile,"Nx=%d; dx=%14.6e; numGhost=%d; n1a=%d; n1b=%d; nd1a=%d; nd1b=%d;\n",Nx,dx,numGhost,ja,jb,nd1a,nd1b);
    fprintf(matlabFile,"solutionName=\'%s\';\n",solutionName);

    if (saveMatlab > 1)
    {
        writeMatlabVector( matlabFile, x_p, "x", nd1a, nd1b );
        writeMatlabVector( matlabFile, u_p[curr], "u", nd1a, nd1b );
        writeMatlabVector( matlabFile, error_p, "err", nd1a, nd1b );
    }

    fclose(matlabFile);
    printf("Wrote file %s\n\n",matlabFileName.c_str());

    delete [] x_p;
    delete [] boundaryCondition_p;
    delete [] u_p[0];
    delete [] u_p[1];
    delete [] error_p;

    hipFree(xa_d); 
    hipFree(nt_d); 
    hipFree(n1a_d); hipFree(n1b_d); hipFree(nd1a_d); hipFree(nd1b_d); hipFree(nd1_d);
    hipFree(x_d); 
    hipFree(u_d[0]); hipFree(u_d[1]);
    hipFree(t_d); hipFree(dt_d); hipFree(rx_d); 

    return 0;
}