#include "hip/hip_runtime.h"
#include <stdio.h>
#include "parseCommand.h"
typedef double Real;

__global__ void add( Real *a_d, Real *b_d, Real *c_d, int n, int Nt ){
	   int index = threadIdx.x + blockIdx.x*Nt;
	   if (index < n){
	      a_d[index] = -index;
	      b_d[index] = index*index;
	      c_d[index] = a_d[index] + b_d[index];
	   }
}

int main( int argc, char *argv[] ) {

    int n=1e+04;;
    int Nt=1;
    string line;
    bool echo = true;
    for( int i=1; i<argc; i++){
      line = argv[i];
      if (parseCommand(line, "-n=", n, echo)) {}
      else if(parseCommand(line, "-Nt=", Nt, echo)) {}
    }
    Real *a_p = new Real [n];
    Real *b_p = new Real [n];
    Real *c_p = new Real [n];

    // serial version
    for( int i=0; i<n; i++ ){
        a_p[i] = -i;
	b_p[i] = i*i;
	c_p[i] = a_p[i] + b_p[i];
    }

    // cuda version
    int Nb = ceil( (1.*n)/Nt);
    Real *a_d;
    Real *b_d;
    Real *c_d;
    int  *n_d;

    // cuda allocate memory
    hipMalloc((void**)&a_d, n*sizeof(Real));
    hipMalloc((void**)&b_d, n*sizeof(Real));
    hipMalloc((void**)&c_d, n*sizeof(Real));
    hipMalloc((void**)&n_d, sizeof(int));

    // cuda copy memory
    hipMemcpy(a_d, a_p, n*sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_p, n*sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(c_d, c_p, n*sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(n_d, &n, sizeof(int), hipMemcpyHostToDevice);

    add<<<Nb, Nt>>>(a_d, b_d, c_d, n, Nt);

    hipMemcpy(a_p, a_d, n*sizeof(Real), hipMemcpyDeviceToHost);
    hipMemcpy(b_p, b_d, n*sizeof(Real), hipMemcpyDeviceToHost);
    hipMemcpy(c_p, c_d, n*sizeof(Real), hipMemcpyDeviceToHost);

    return 0;
}